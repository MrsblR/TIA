#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <chrono>
#include <random> 

__global__ void sumVectorsKernel(float* vA, float* vB, float* result, int n){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) result[i] = vA[i] + vB[i];
}

int main(){
    int n = 1000000;
    size_t size = n * sizeof(float);

    float* vA = (float*)malloc(size);
    float* vB = (float*)malloc(size);
    float* result = (float*)malloc(size);

    std::random_device rd;
    std::mt19937 generator(rd());
    std::uniform_real_distribution<float> distribution(0.0f, 100.0f);

    for (int i = 0; i < n; i++){
        vA[i] = distribution(generator);
        vB[i] = distribution(generator);
    }

    float* d_vA, * d_vB, * d_result;
    hipMalloc((void**)&d_vA, size);
    hipMalloc((void**)&d_vB, size);
    hipMalloc((void**)&d_result, size);

    hipMemcpy(d_vA, vA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_vB, vB, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    auto cpu_start = std::chrono::high_resolution_clock::now();
    hipEventRecord(start);

    sumVectorsKernel <<<numBlocks, blockSize >>> (d_vA, d_vB, d_result, n);

    hipEventRecord(stop);

    hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);

    auto cpu_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> cpu_duration = cpu_end - cpu_start;

    float gpu_milliseconds = 0;
    hipEventElapsedTime(&gpu_milliseconds, start, stop);

    bool verification = true;
    for (int i = 0; i < 10; i++)
        if (result[i] != vA[i] + vB[i]){
            printf("Error en Elemento[%d] = %f\n", i, result[i]);
            verification = false;
            break;
        }

    if (verification){
        printf("Listo\n");
    }

    printf("Tiempo de ejecucion del kernel en la GPU: %f ms\n", gpu_milliseconds);
    printf("Tiempo total de ejecucion en la GPU: %f ms\n", cpu_duration.count());

    hipFree(d_vA);
    hipFree(d_vB);
    hipFree(d_result);

    free(vA);
    free(vB);
    free(result);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}